#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#include <math.h>

#define SIZE  320  // Define image size as 320x320
#define B 256;     // Define block size
#define TH  256;   // Define the number of threads
#define TOTAL 256*256  // Define the total number of threads

// Declare GPU and CPU convolution functions
__global__ void CNN_GPU(int m1[], unsigned char m2[], int kernel[], int size_, int num, int big);
void CNN_CPU(int* m1, unsigned char *m2, int kernel[], int size_, int num, int big);

int main(int argc, char **argv) {
    // Define Sobel and Gaussian convolution kernels
    int kernel1[9] = { 1,0,-1,2,0,-2,3,0,-3 };
    int kernel2[25] = {1,4,6,4,1,4,16,24,16,4,6,24,36,24,6,4,16,24,16,4,1,4,6,4,1};	

    clock_t  start, end, S_t, G_t;  // Timing variables
    FILE* bin;
    bin = fopen("/home/a1075501/convolution/1280.bin", "rb"); // Open binary image file
    unsigned char c;
    
    // Allocate memory for image data
    int* matrix = (int*)malloc(sizeof(int) * SIZE * SIZE);
    unsigned char* s_result = (unsigned char*)malloc(sizeof(int) * (SIZE-2) * (SIZE-2));
    unsigned char* g_result = (unsigned char*)malloc(sizeof(int) * (SIZE-4) * (SIZE-4));
    
    // Read image data into matrix
    if (bin) {
        int i = 0;
        while ((c = fgetc(bin)) != EOF) {
            matrix[i++] = int(c);
            if (i == SIZE * SIZE)
                break;
        }
    }
    fclose(bin);
    
    // Perform Sobel convolution on CPU
    start = clock();
    CNN_CPU(matrix, s_result, kernel1, SIZE, 1, 3);
    end = clock();
    S_t = end - start;
    printf("CPU_sobel: %lf s \n", (double) S_t / CLOCKS_PER_SEC);
    free(s_result);
    
    // Perform Gaussian convolution on CPU
    start = clock();
    CNN_CPU(matrix, g_result, kernel2, SIZE-2, 256, 5);
    end = clock();
    G_t = end - start;
    printf("CPU_gaussian: %lf s \n", (double) G_t / CLOCKS_PER_SEC);
    free(g_result);

    // Initialize CUDA timers
    hipEvent_t c_start, c_stop;
    hipEventCreate(&c_start);
    hipEventCreate(&c_stop);

    // Allocate memory on GPU
    int *matrix_g;
    unsigned char *G_s_result, *G_g_result;
    int *kernel1_G, *kernel2_G;
    float GTime, STime = 0;

    hipMalloc((void**)&matrix_g, sizeof(int) * SIZE * SIZE);
    hipMalloc((void**)&G_s_result, sizeof(int) * (SIZE-2) * (SIZE-2));
    hipMalloc((void**)&G_g_result, sizeof(int) * (SIZE-4) * (SIZE-4));
    hipMalloc((void**)&kernel1_G, sizeof(int) * 9);
    hipMalloc((void**)&kernel2_G, sizeof(int) * 25);

    // Allocate memory on CPU
    s_result = (unsigned char*)malloc(sizeof(int) * (SIZE-2) * (SIZE-2));
    g_result = (unsigned char*)malloc(sizeof(int) * (SIZE-4) * (SIZE-4));

    // Copy data to GPU
    hipMemcpy(matrix_g, matrix, sizeof(int) * SIZE * SIZE, hipMemcpyHostToDevice);
    hipMemcpy(kernel1_G, kernel1, sizeof(int) * 9, hipMemcpyHostToDevice);
    hipMemcpy(kernel2_G, kernel2, sizeof(int) * 25, hipMemcpyHostToDevice);

    // Perform Sobel convolution on GPU
    hipEventRecord(c_start, 0);
    CNN_GPU<<<256, 256>>>(matrix_g, G_s_result, kernel1_G, SIZE, 1, 9);
    hipEventRecord(c_stop, 0);
    hipEventSynchronize(c_stop);
    hipEventElapsedTime(&STime, c_start, c_stop);
    hipMemcpy(s_result, G_s_result, sizeof(int) * (SIZE-2) * (SIZE-2), hipMemcpyDeviceToHost);
    printf("GPU_sobel: %lf s \n", (double) STime / 1000);
    free(s_result);
    hipFree(kernel1_G);
    hipFree(G_s_result);

    // Perform Gaussian convolution on GPU
    hipEventRecord(c_start, 0);
    CNN_GPU<<<256, 256>>>(matrix_g, G_g_result, kernel2_G, SIZE-2, 256, 25);
    hipEventRecord(c_stop, 0);
    hipEventSynchronize(c_stop);
    hipEventElapsedTime(&GTime, c_start, c_stop);
    hipMemcpy(g_result, G_g_result, sizeof(int) * (SIZE-2) * (SIZE-2), hipMemcpyDeviceToHost);
    printf("GPU_gaussian %lf s \n", (double) GTime / 1000);

    // Free memory
    free(g_result);
    hipFree(G_g_result);
    hipFree(kernel2_G);
    hipFree(matrix_g);
    free(matrix);

    return 0;
}

// CPU convolution function
void CNN_CPU(int *m1, unsigned char *m2, int kernel[], int size_, int num, int big) {
    for (int i = 0; i < size_ - 2; i++) {
        for (int j = 0; j < size_ - 2; j++) {
            int t = 0;
            for (int k = 0; k < big * big; k++) {
                t = t + kernel[k] * m1[(k / big + i) * SIZE + k % big + j];
            }
            t /= num;
            if (t < 0) t = 0;
            else if (t > 255) t = 255;
            m2[i * (size_ - 2) + j] = (unsigned char)t;
        }
    }
}

// GPU convolution kernel
__global__ void CNN_GPU(int m1[], unsigned char m2[], int kernel[], int size_, int num, int big) {
    int bias, temp_B;
    if (big == 9) {
        bias = size_;
        temp_B = 3;
    } else if (big == 25) {
        bias = size_ - 2;
        temp_B = 5;
    }

    int bb = bias * bias;
    int j = blockDim.x * blockIdx.x + threadIdx.x;

    for (int i = 0; i < bb; i += TOTAL) {
        if (i + j < bb) {
            int t = 0;
            for (int k = 0; k < big; k++) {
                t += kernel[k] * m1[(k / temp_B + (i + j) / bias) * size_ + k % temp_B + (i + j) % bias];
            }
            t /= num;
            if (t < 0) t = 0;
            else if (t > 255) t = 255;
            m2[(i + j) / bias * bias + (i + j) % bias] = (unsigned char)t;
        }
    }
}
